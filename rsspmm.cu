#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

int IN_TILE_ROW_SIZE = 32;
int IN_TILE_SLICE_SIZE = 32;
int WARP_SIZE = 32;
int tid = threadIdx.x;
int tb_idx = blockIdx.x;
int start = 0;
int end = 0;
int row_offset;
int slice_offset;
int warp_id;
int lane_id;
int i;
[int][int] sm_input_value[IN_TILE_ROW_SIZE][WARP_SIZE];
[int][int] input_value[IN_TILE_ROW_SIZE][IN_TILE_SLICE_SIZE];
int seg_start_num[IN_TILE_ROW_SIZE];
int start_seg_position[IN_TILE_ROW_SIZE];
int index_buf;
int value_buf;
int val;
int mod;
int row_idx;
int seg_index[IN_TILE_ROW_SIZE];


//Heavy Row Segments
void heavy(){
	row_offset = tb_idx * IN_TILE_ROW_SIZE ;
	slice_offset = tb_idy * IN_TILE_SLICE_SIZE ;
	warp_id = tid / WARP_SIZE ;
	lane_id = tid % WARP_SIZE ;

	for i = warp_id to IN_TILE_ROW_SIZE step tb . size ()/ WARP_SIZE do
		sm_input_value [ i ][ lane_id ] = input_value [ row_offset +i ][ slice_offset + lane_id ];
	end

	__syncthreads ;

	for i = seg_start_num [ tb_idx ] to seg_start_num [ tb_idx +1] -1 step tb.size ()/ WARP_SIZE do
		val = 0;
		start = start_seg_position [i ];
		end = start_seg_position [i +1];

		for j = start to end -1 do
			mod = ( j - start )% WARP_SIZE

			if mod == 0 then
				index_buf = seg_index [ j + lane_id ];
				value_buf = seg_value [ j + lane_id ];
			end
			val += sm_input_value [ __shfl ( index_buf , mod )][ lane_id ]
			* __shfl ( value_buf , mod );
		end
		
		row_idx = seg_row_position [i ];
		// directly accumulate results in global memory
		atomicAdd (& dest_value [ row_idx ][ slice_offset + lane_id ] , val );
	end
}




//light rows 
void light(){

	row_offset = ( tb_idx * tb . size () + tid ) / WARP_SIZE ;
	slice_offset = tb_idy * IN_TILE_COL_SIZE ;
	lane_id = tid % WARP_SIZE ;
	start = csr_row_pointer [ row_offset ];
	end = csr_row_pointer [ row_offset +1];
	val = 0;
	for i = start to end -1 do
		mod = ( i - start )% WARP_SIZE
		if mod == 0 then
			index_buf = csr_column_idx [ i + lane_id ];
			value_buf = csr_column_val [ i + lane_id ];
		end
		val += input_value [ __shfl ( index_buf , mod )][ lane_id ]
		* __shfl ( value_buf , mod );
	end
	// directly accumulate results in global memory
	atomicAdd (& dest_value [ row_offset ][ slice_offset + lane_id ] , val );
}
